#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#define NUM_THREADS 500


template <typename scalar_t>
__global__ void get_faces_vertices_kernel
   (torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> array,
    int C,
    int H,
    int W,
    torch::PackedTensorAccessor<scalar_t,1,torch::RestrictPtrTraits,size_t> threshold,
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> result1,
    torch::PackedTensorAccessor<scalar_t,3,torch::RestrictPtrTraits,size_t> result2) {

    auto t = blockDim.x * blockIdx.x + threadIdx.x;
    const auto per_thread = (C * H * W) / NUM_THREADS;
    t *= per_thread;

    if (t < C * H * W) {
        for (int k = t; k < t + per_thread; k++) {
            int channel = k % C;
            int i = k / C % H;
            int j = k / C / H % W;

            int v_cell = 0;
            int f_cell = 0;

            scalar_t f1 = (scalar_t)channel;
            scalar_t f2 = (scalar_t)i;
            scalar_t f3 = (scalar_t)j;

            if (array[channel][i][j] > threshold[0]) {
                if (channel == 0 || array[channel - 1][i][j] <= threshold[0]) {
                     result1[k][v_cell+0][0]=(f1-0.5);
                     result1[k][v_cell+0][1]=(f2-0.5);
                     result1[k][v_cell+0][2]=(f3-0.5);


                     result1[k][v_cell+1][0]=(f1-0.5);
                     result1[k][v_cell+1][1]=(f2-0.5);
                     result1[k][v_cell+1][2]=(f3+0.5);

                     result1[k][v_cell+2][0]=(f1-0.5);
                     result1[k][v_cell+2][1]=(f2+0.5);
                     result1[k][v_cell+2][2]=(f3-0.5);

                     result1[k][v_cell+3][0]=(f1-0.5);
                     result1[k][v_cell+3][1]=(f2+0.5);
                     result1[k][v_cell+3][2]=(f3+0.5);

                     result2[k][f_cell+0][0]=result1[k][v_cell+0][0];
                     result2[k][f_cell+0][1]=result1[k][v_cell+0][1];
                     result2[k][f_cell+0][2]=result1[k][v_cell+0][2];
                     result2[k][f_cell+0][3]=result1[k][v_cell+1][0];
                     result2[k][f_cell+0][4]=result1[k][v_cell+1][1];
                     result2[k][f_cell+0][5]=result1[k][v_cell+1][2];
                     result2[k][f_cell+0][6]=result1[k][v_cell+2][0];
                     result2[k][f_cell+0][7]=result1[k][v_cell+2][1];
                     result2[k][f_cell+0][8]=result1[k][v_cell+2][2];

                     result2[k][f_cell+1][0]=result1[k][v_cell+1][0];
                     result2[k][f_cell+1][1]=result1[k][v_cell+1][1];
                     result2[k][f_cell+1][2]=result1[k][v_cell+1][2];
                     result2[k][f_cell+1][3]=result1[k][v_cell+2][0];
                     result2[k][f_cell+1][4]=result1[k][v_cell+2][1];
                     result2[k][f_cell+1][5]=result1[k][v_cell+2][2];
                     result2[k][f_cell+1][6]=result1[k][v_cell+3][0];
                     result2[k][f_cell+1][7]=result1[k][v_cell+3][1];
                     result2[k][f_cell+1][8]=result1[k][v_cell+3][2];

                     v_cell = v_cell + 4;
                     f_cell = f_cell + 2;
                }
                if (channel == C - 1 || array[channel + 1][i][j] <= threshold[0]) {
                     result1[k][v_cell+0][0]=(f1+0.5);
                     result1[k][v_cell+0][1]=(f2-0.5);
                     result1[k][v_cell+0][2]=(f3-0.5);
                     result1[k][v_cell+1][0]=(f1+0.5);
                     result1[k][v_cell+1][1]=(f2-0.5);
                     result1[k][v_cell+1][2]=(f3+0.5);
                     result1[k][v_cell+2][0]=(f1+0.5);
                     result1[k][v_cell+2][1]=(f2+0.5);
                     result1[k][v_cell+2][2]=(f3-0.5);
                     result1[k][v_cell+3][0]=(f1+0.5);
                     result1[k][v_cell+3][1]=(f2+0.5);
                     result1[k][v_cell+3][2]=(f3+0.5);

                     result2[k][f_cell+0][0]=result1[k][v_cell+0][0];
                     result2[k][f_cell+0][1]=result1[k][v_cell+0][1];
                     result2[k][f_cell+0][2]=result1[k][v_cell+0][2];
                     result2[k][f_cell+0][3]=result1[k][v_cell+1][0];
                     result2[k][f_cell+0][4]=result1[k][v_cell+1][1];
                     result2[k][f_cell+0][5]=result1[k][v_cell+1][2];
                     result2[k][f_cell+0][6]=result1[k][v_cell+2][0];
                     result2[k][f_cell+0][7]=result1[k][v_cell+2][1];
                     result2[k][f_cell+0][8]=result1[k][v_cell+2][2];

                     result2[k][f_cell+1][0]=result1[k][v_cell+1][0];
                     result2[k][f_cell+1][1]=result1[k][v_cell+1][1];
                     result2[k][f_cell+1][2]=result1[k][v_cell+1][2];
                     result2[k][f_cell+1][3]=result1[k][v_cell+2][0];
                     result2[k][f_cell+1][4]=result1[k][v_cell+2][1];
                     result2[k][f_cell+1][5]=result1[k][v_cell+2][2];
                     result2[k][f_cell+1][6]=result1[k][v_cell+3][0];
                     result2[k][f_cell+1][7]=result1[k][v_cell+3][1];
                     result2[k][f_cell+1][8]=result1[k][v_cell+3][2];

                     v_cell = v_cell + 4;
                     f_cell = f_cell + 2;
                }
                if (i == 0 || array[channel][i - 1][j] <= threshold[0]) {
                     result1[k][v_cell+0][0]=(f1+0.5);
                     result1[k][v_cell+0][1]=(f2-0.5);
                     result1[k][v_cell+0][2]=(f3-0.5);
                     result1[k][v_cell+1][0]=(f1+0.5);
                     result1[k][v_cell+1][1]=(f2-0.5);
                     result1[k][v_cell+1][2]=(f3+0.5);
                     result1[k][v_cell+2][0]=(f1-0.5);
                     result1[k][v_cell+2][1]=(f2-0.5);
                     result1[k][v_cell+2][2]=(f3-0.5);
                     result1[k][v_cell+3][0]=(f1-0.5);
                     result1[k][v_cell+3][1]=(f2-0.5);
                     result1[k][v_cell+3][2]=(f3+0.5);

                     result2[k][f_cell+0][0]=result1[k][v_cell+0][0];
                     result2[k][f_cell+0][1]=result1[k][v_cell+0][1];
                     result2[k][f_cell+0][2]=result1[k][v_cell+0][2];
                     result2[k][f_cell+0][3]=result1[k][v_cell+1][0];
                     result2[k][f_cell+0][4]=result1[k][v_cell+1][1];
                     result2[k][f_cell+0][5]=result1[k][v_cell+1][2];
                     result2[k][f_cell+0][6]=result1[k][v_cell+2][0];
                     result2[k][f_cell+0][7]=result1[k][v_cell+2][1];
                     result2[k][f_cell+0][8]=result1[k][v_cell+2][2];

                     result2[k][f_cell+1][0]=result1[k][v_cell+1][0];
                     result2[k][f_cell+1][1]=result1[k][v_cell+1][1];
                     result2[k][f_cell+1][2]=result1[k][v_cell+1][2];
                     result2[k][f_cell+1][3]=result1[k][v_cell+2][0];
                     result2[k][f_cell+1][4]=result1[k][v_cell+2][1];
                     result2[k][f_cell+1][5]=result1[k][v_cell+2][2];
                     result2[k][f_cell+1][6]=result1[k][v_cell+3][0];
                     result2[k][f_cell+1][7]=result1[k][v_cell+3][1];
                     result2[k][f_cell+1][8]=result1[k][v_cell+3][2];

                     v_cell = v_cell + 4;
                     f_cell = f_cell + 2;
                }
                if (i == H - 1 || array[channel][i + 1][j] <= threshold[0]) {
                     result1[k][v_cell+0][0]=(f1-0.5);
                     result1[k][v_cell+0][1]=(f2+0.5);
                     result1[k][v_cell+0][2]=(f3-0.5);
                     result1[k][v_cell+1][0]=(f1-0.5);
                     result1[k][v_cell+1][1]=(f2+0.5);
                     result1[k][v_cell+1][2]=(f3+0.5);
                     result1[k][v_cell+2][0]=(f1+0.5);
                     result1[k][v_cell+2][1]=(f2+0.5);
                     result1[k][v_cell+2][2]=(f3-0.5);
                     result1[k][v_cell+3][0]=(f1+0.5);
                     result1[k][v_cell+3][1]=(f2+0.5);
                     result1[k][v_cell+3][0]=(f3+0.5);

                     result2[k][f_cell+0][0]=result1[k][v_cell+0][0];
                     result2[k][f_cell+0][1]=result1[k][v_cell+0][1];
                     result2[k][f_cell+0][2]=result1[k][v_cell+0][2];
                     result2[k][f_cell+0][3]=result1[k][v_cell+1][0];
                     result2[k][f_cell+0][4]=result1[k][v_cell+1][1];
                     result2[k][f_cell+0][5]=result1[k][v_cell+1][2];
                     result2[k][f_cell+0][6]=result1[k][v_cell+2][0];
                     result2[k][f_cell+0][7]=result1[k][v_cell+2][1];
                     result2[k][f_cell+0][8]=result1[k][v_cell+2][2];

                     result2[k][f_cell+1][0]=result1[k][v_cell+1][0];
                     result2[k][f_cell+1][1]=result1[k][v_cell+1][1];
                     result2[k][f_cell+1][2]=result1[k][v_cell+1][2];
                     result2[k][f_cell+1][3]=result1[k][v_cell+2][0];
                     result2[k][f_cell+1][4]=result1[k][v_cell+2][1];
                     result2[k][f_cell+1][5]=result1[k][v_cell+2][2];
                     result2[k][f_cell+1][6]=result1[k][v_cell+3][0];
                     result2[k][f_cell+1][7]=result1[k][v_cell+3][1];
                     result2[k][f_cell+1][8]=result1[k][v_cell+3][2];

                     v_cell = v_cell + 4;
                     f_cell = f_cell + 2;
                }
                if (j == 0 || array[channel][i][j - 1] <= threshold[0]) {
                     result1[k][v_cell+0][0]=(f1+0.5);
                     result1[k][v_cell+0][1]=(f2-0.5);
                     result1[k][v_cell+0][2]=(f3-0.5);
                     result1[k][v_cell+1][0]=(f1-0.5);
                     result1[k][v_cell+1][1]=(f2-0.5);
                     result1[k][v_cell+1][2]=(f3-0.5);
                     result1[k][v_cell+2][0]=(f1+0.5);
                     result1[k][v_cell+2][1]=(f2+0.5);
                     result1[k][v_cell+2][2]=(f3-0.5);
                     result1[k][v_cell+3][0]=(f1-0.5);
                     result1[k][v_cell+3][1]=(f2+0.5);
                     result1[k][v_cell+3][2]=(f3-0.5);

                     result2[k][f_cell+0][0]=result1[k][v_cell+0][0];
                     result2[k][f_cell+0][1]=result1[k][v_cell+0][1];
                     result2[k][f_cell+0][2]=result1[k][v_cell+0][2];
                     result2[k][f_cell+0][3]=result1[k][v_cell+1][0];
                     result2[k][f_cell+0][4]=result1[k][v_cell+1][1];
                     result2[k][f_cell+0][5]=result1[k][v_cell+1][2];
                     result2[k][f_cell+0][6]=result1[k][v_cell+2][0];
                     result2[k][f_cell+0][7]=result1[k][v_cell+2][1];
                     result2[k][f_cell+0][8]=result1[k][v_cell+2][2];

                     result2[k][f_cell+1][0]=result1[k][v_cell+1][0];
                     result2[k][f_cell+1][1]=result1[k][v_cell+1][1];
                     result2[k][f_cell+1][2]=result1[k][v_cell+1][2];
                     result2[k][f_cell+1][3]=result1[k][v_cell+2][0];
                     result2[k][f_cell+1][4]=result1[k][v_cell+2][1];
                     result2[k][f_cell+1][5]=result1[k][v_cell+2][2];
                     result2[k][f_cell+1][6]=result1[k][v_cell+3][0];
                     result2[k][f_cell+1][7]=result1[k][v_cell+3][1];
                     result2[k][f_cell+1][8]=result1[k][v_cell+3][2];

                     v_cell = v_cell + 4;
                     f_cell = f_cell + 2;
                }
                if (j == W - 1 || array[channel][i][j + 1] <= threshold[0]) {
                     result1[k][v_cell+0][0]=(f1-0.5);
                     result1[k][v_cell+0][1]=(f2-0.5);
                     result1[k][v_cell+0][2]=(f3+0.5);
                     result1[k][v_cell+1][0]=(f1+0.5);
                     result1[k][v_cell+1][1]=(f2-0.5);
                     result1[k][v_cell+1][2]=(f3+0.5);
                     result1[k][v_cell+2][0]=(f1-0.5);
                     result1[k][v_cell+2][1]=(f2+0.5);
                     result1[k][v_cell+2][2]=(f3+0.5);
                     result1[k][v_cell+3][0]=(f1+0.5);
                     result1[k][v_cell+3][1]=(f2+0.5);
                     result1[k][v_cell+3][2]=(f3+0.5);

                     result2[k][f_cell+0][0]=result1[k][v_cell+0][0];
                     result2[k][f_cell+0][1]=result1[k][v_cell+0][1];
                     result2[k][f_cell+0][2]=result1[k][v_cell+0][2];
                     result2[k][f_cell+0][3]=result1[k][v_cell+1][0];
                     result2[k][f_cell+0][4]=result1[k][v_cell+1][1];
                     result2[k][f_cell+0][5]=result1[k][v_cell+1][2];
                     result2[k][f_cell+0][6]=result1[k][v_cell+2][0];
                     result2[k][f_cell+0][7]=result1[k][v_cell+2][1];
                     result2[k][f_cell+0][8]=result1[k][v_cell+2][2];

                     result2[k][f_cell+1][0]=result1[k][v_cell+1][0];
                     result2[k][f_cell+1][1]=result1[k][v_cell+1][1];
                     result2[k][f_cell+1][2]=result1[k][v_cell+1][2];
                     result2[k][f_cell+1][3]=result1[k][v_cell+2][0];
                     result2[k][f_cell+1][4]=result1[k][v_cell+2][1];
                     result2[k][f_cell+1][5]=result1[k][v_cell+2][2];
                     result2[k][f_cell+1][6]=result1[k][v_cell+3][0];
                     result2[k][f_cell+1][7]=result1[k][v_cell+3][1];
                     result2[k][f_cell+1][8]=result1[k][v_cell+3][2];

                     v_cell = v_cell + 4;
                     f_cell = f_cell + 2;
                }

            }
        }

    }

}



void get_faces_vertices_cuda(torch::Tensor voxel, int  C, int H
            ,int W,torch::Tensor threshold ,torch::Tensor result1, torch::Tensor result2){

  const int threads = NUM_THREADS;
  const int blocks = (threads/1024)+1;

  AT_DISPATCH_FLOATING_TYPES(voxel.type(), "get_faces_vertices_cuda", ([&] {
    get_faces_vertices_kernel<scalar_t><<<blocks, threads>>>(
        voxel.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
        C,
        H,
        W,
        threshold.packed_accessor<scalar_t,1,torch::RestrictPtrTraits,size_t>(),
        result1.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>(),
        result2.packed_accessor<scalar_t,3,torch::RestrictPtrTraits,size_t>());
  }));

}